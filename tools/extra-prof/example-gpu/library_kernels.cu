#include "hip/hip_runtime.h"
#include "library.hpp"
#include <hip/hip_runtime.h>
#include <>
#include <stddef.h>
#include <stdio.h>

__global__ void gpuKernelLib(int n, float a, float* x, float* y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        y[i] = a * x[i] + y[i];
}

void runDynamicKernel() {
    int N = 1 << 10;
    float *x, *y, *d_x, *d_y;
    x = new float[N];
    y = new float[N];

    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    gpuKernelLib<<<10, 1024>>>(N, 2.0f, d_x, d_y);

    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = max(maxError, abs(y[i] - 4.0f));
    printf("Max error: %f\n", maxError);

    hipFree(d_x);
    hipFree(d_y);
    delete[] x;
    delete[] y;
}